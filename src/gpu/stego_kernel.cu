#include "hip/hip_runtime.h"
#include "config.h"
#include <hip/hip_complex.h>

extern "C" __global__ void embed_kernel(float *cover, float *secret, float *stego, float alpha, int length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        stego[idx] = cover[idx] + alpha * secret[idx];
    }
}
